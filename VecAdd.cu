#include "hip/hip_runtime.h"
// Kernel definition

__global__ void VecAdd(float* A, float* B, float* C)

{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

int main()
{
	const int N  = 10;
	double A[], B[], C[];
	
	__global__ void VecAdd( A, B, C);
	
	// Kernel invoaction with N threads
	VecAdd<<<1, N>>>(A,B,C);
	
}

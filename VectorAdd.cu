
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;

	if(i<n)
		c[i] = a[i] + b[i];
}

int main()
{
	printf("\n------------------------------\nSUMA VECTORIAL\n------------------------------\n");
// ALLOCATE AND INITIALIZE DATA ON CPU
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));

// ALOCATE DATA ON GPU
	hipMalloc( &d_a, SIZE*sizeof(int));
	hipMalloc( &d_b, SIZE*sizeof(int));
	hipMalloc( &d_c, SIZE*sizeof(int));

	for( int i=0;i<SIZE;++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	printf("\n Vector A\n \n");
	for( int i=0;i<3;++i)
		printf("a[%d] = %d\n", i, a[i]);

	printf(".\n.\n.\n");

	for( int i=SIZE-3;i<SIZE;++i)
		printf("a[%d] = %d\n", i, a[i]);

	printf("\n Vector B\n \n");
	for( int i=0;i<3;++i)
		printf("b[%d] = %d\n", i, b[i]);

	printf(".\n.\n.\n");

	for( int i=SIZE-3;i<SIZE;++i)
		printf("b[%d] = %d\n", i, b[i]);


// TRANSFER DATA FROM CPU TO GPU
	hipMemcpy( d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice );

// RUN KERNEL
	VectorAdd<<< 1, SIZE >>>(d_a, d_b, d_c, SIZE);

// TRANSFER DATA FROM GPU TO CPU
	hipMemcpy( c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost );

	printf("\n A[i] + B[i] = C[i] \n");

	printf("\n Vector C\n \n");
	for( int i=0;i<3;++i)
		printf("c[%d] = %d\n", i, c[i]);

	printf(".\n.\n.\n");

	for( int i=SIZE-3;i<SIZE;++i)
		printf("c[%d] = %d\n", i, c[i]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

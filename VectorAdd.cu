
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;

	if(i<n)
		c[i] = a[i] + b[i];
}

int main()
{
// ALLOCATE AND INITIALIZE DATA ON CPU
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));

// ALOCATE DATA ON GPU
	hipMalloc( &d_a, SIZE*sizeof(int));
	hipMalloc( &d_b, SIZE*sizeof(int));
	hipMalloc( &d_c, SIZE*sizeof(int));

	for( int i=0;i<SIZE;++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	printf("\n Vector A\n \n");
	for( int i=0;i<10;++i)
		printf("a[%d] = %d\n", i, a[i]);

	printf("\n Vector B\n \n");
	for( int i=0;i<10;++i)
		printf("b[%d] = %d\n", i, b[i]);

// TRANSFER DATA FROM CPU TO GPU
	hipMemcpy( d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice );

// RUN KERNEL
	VectorAdd<<< 1, SIZE >>>(d_a, d_b, d_c, SIZE);

// TRANSFER DATA FROM GPU TO CPU
	hipMemcpy( c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost );

	printf("\n A[i] + B[i] = C[i] , i=0,1, ... ,9 \n");

	printf("\n Vector C\n \n");
	for( int i=0;i<10;++i)
		printf("c[%d] = %d\n", i, c[i]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
